#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/random.h>
#include <thrust/functional.h>

using namespace thrust::placeholders;

struct saxpy_functor
{
    const int a;

    saxpy_functor(int _a) : a(_a) {}

    __host__ __device__ int operator()(const int &b) const
    {
        return a + b;
    }
} functorx;
/*
void fast_fast_fast(int A, thrust::device_vector<int>& B)
{
    // B <- A  + B
    thrust::transform(B.begin(), B.begin(), saxpy_functor(A));
}
*/
/*
template <typename T = void>
struct myplus
{
    typedef T first_argument_type;
    typedef T second_argument_type;
    typedef T result_type;
    __thrust_exec_check_disable__
        __host__ __device__ constexpr T
        operator()(const T &lhs, const T &rhs) const
    {
        return lhs + rhs;
    }
};
*/
int main(int argc, char **argv)
{
    double timer, timer_alloc;
    clock_t start, start_alloc, stop;
    if (argc < 2)
    {
        std::cerr << "You should put second argument as log_2(N)" << std::endl;
        return 1;
    }

    int logN = atoi(argv[1]);
    logN = logN < 4 || logN > 30 ? 4 : logN;
    int N = 1 << logN;

    thrust::host_vector<int> h(1 << logN);
    thrust::generate(h.begin(), h.end(), rand);

    thrust::device_vector<int> a(1 << logN);
    thrust::device_vector<int> b(1 << logN);

    for (int i = 0; i < N; i += N / 16)
    {
        std::cout << h[i] << std::endl;
    }

    start_alloc = clock();

    thrust::copy(h.begin(), h.end(), a.begin());
    thrust::copy(h.begin(), h.end(), b.begin());
    thrust::sort(b.begin(), b.end());

    // for (int i = 0; i < logN; i++) {
    start = clock();
    thrust::transform(a.begin(), a.end(), b.begin(), b.begin(), thrust::plus<int>());
    thrust::transform(a.begin(), a.end(), b.begin(), b.begin(), functorx);
    // }

    stop = clock();
    timer = 1000 * ((double)(stop - start)) / (double)CLOCKS_PER_SEC;
    timer_alloc = 1000 * ((double)(stop - start_alloc)) / (double)CLOCKS_PER_SEC;

    std::cout << std::endl;
    thrust::copy(a.begin(), a.end(), h.begin());

    for (int i = 0; i < N; i += N / 16)
    {
        std::cout << h[i] << std::endl;
    }

    std::cout << "Thrust Elapsed time: " << timer << " ms (" << timer_alloc << " ms)" << std::endl;
}