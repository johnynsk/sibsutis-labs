
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <malloc.h>
#include <time.h>

__global__ void gInit(int *a, int *b)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  a[i] = 2 * i;
  b[i] = 2 * i + 1;
}

__global__ void gSum(int *a, int *b)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  a[i] += b[i];
}

// a[i] := 2*i + (b[i]) := 2*i + 2*i + 1 = 4*i + 1;

int main(int argc, char **argv)
{
  if (argc < 3)
  {
    fprintf(stderr, "USAGE: <prog> <log_2{N}> <num_threads>");
    return -1;
  }

  double timer, timer_alloc;
  clock_t start, start_alloc, stop;
  hipEvent_t startE, stopE;
  hipEventCreate(&startE);
  hipEventCreate(&stopE);
  float gpuTime = 0.0f;

  int N = 1 << atoi(argv[1]);
  if (N == 1)
    N = 30720;
  int num_threads = atoi(argv[2]);
  int num_blocks = N / num_threads;

  int *a, *b;
  int *a_h;

  start_alloc = clock();
  hipMalloc((void **)&a, N * sizeof(int));
  hipMalloc((void **)&b, N * sizeof(int));
  a_h = (int *)calloc(N, sizeof(int));

  gInit<<<num_blocks, num_threads>>>(a, b);
  hipDeviceSynchronize();

  hipEventRecord(startE, 0);
  start = clock();
  gSum<<<num_blocks, num_threads>>>(a, b);
  hipDeviceSynchronize();
  hipEventRecord(stopE, 0);
  hipEventSynchronize(stopE);
  hipEventElapsedTime(&gpuTime, startE, stopE);

  hipMemcpy(a_h, a, N * sizeof(int), hipMemcpyDeviceToHost);

  stop = clock();
  timer = 1000 * ((double)(stop - start)) / (double)CLOCKS_PER_SEC;
  timer_alloc = 1000 * ((double)(stop - start_alloc)) / (double)CLOCKS_PER_SEC;

  // for (int i = 0; i < N; i += N / 16)
  // fprintf(stdout, "%d\n", a_h[i]);

  fprintf(stdout, "CUDA Elapsed time: %g ms (%g ms)\n", gpuTime, timer_alloc);

  hipFree(a);
  hipFree(b);
  free(a_h);

  return 0;
}
