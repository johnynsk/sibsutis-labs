#include "hip/hip_runtime.h"
  #include <thrust/host_vector.h>
  #include <thrust/device_vector.h>
  #include <thrust/generate.h>
  #include <thrust/sort.h>
  #include <thrust/copy.h>
  #include <thrust/random.h>
  #include <iostream>
  #include <time.h>

  int main(int argc, char **argv) {
    double timer, timer_alloc;
    clock_t start, start_alloc, stop;

    hipEvent_t startE, stopE;
    float gpuTime = 0.0f;

    int N = atoi(argv[1]);

    hipEventCreate ( &startE );
    hipEventCreate ( &stopE ); 
    
    
    thrust::host_vector<int> h_vec_0(1 << atoi(argv[1]));
    thrust::device_vector<int> d_vec_1(1 << atoi(argv[1]));
    thrust::device_vector<int> d_vec_2(1 << atoi(argv[1]));
    
    start_alloc = clock();

    thrust::generate(thrust::host, h_vec_0.begin(), h_vec_0.end(), rand);
    thrust::copy(h_vec_0.begin(), h_vec_0.end(), d_vec_1.begin());
    thrust::copy(h_vec_0.begin(), h_vec_0.end(), d_vec_2.begin());
    thrust::sort(d_vec_2.begin(), d_vec_2.end());
    
    start = clock();

    hipEventRecord ( startE, 0 );
    thrust::transform(d_vec_1.begin(), d_vec_1.end(), d_vec_2.begin(), d_vec_2.begin(), thrust::plus<int>());
    hipEventRecord ( stopE, 0 );
    hipEventSynchronize ( stopE );
    hipEventElapsedTime ( &gpuTime, startE, stopE );
    stop = clock();
    
    timer = 1000 * ((double)(stop - start)) / (double)CLOCKS_PER_SEC;
    timer_alloc = 1000 * ((double)(stop - start_alloc)) / (double)CLOCKS_PER_SEC;

    thrust::copy(d_vec_2.begin(), d_vec_2.end(), h_vec_0.begin());

    for (int i = 0; i < (1 << N); i += (1 << N) / 16)
    {
        std::cout << h_vec_0[i] << std::endl;
    }
    
    fprintf(stdout, "Thrust Elapsed time: %f ms (%g ms)\n", gpuTime, timer_alloc);

    //td::cout << "time: " << gpuTime << "sec" << std::endl;
  }

